#include "hip/hip_runtime.h"
#include "common_kernels.hpp"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
// Utilities and system includes
#include <hip/hip_runtime_api.h>  // helper function CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples

__device__ double3 operator+(const double3 &a, const double3 &b) {

	return make_double3(a.x + b.x, a.y + b.y, a.z + b.z);

}
__device__ double3 operator-(const double3 &a, const double3 &b) {

	return make_double3(a.x - b.x, a.y - b.y, a.z - b.z);

}


__device__ double dot_product(const double3 &a, const double3 &b) {

	return (a.x * b.x + a.y * b.y + a.z * b.z);

}


__device__ double myatomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val +
				__longlong_as_double(assumed)));

		// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}

__global__ void add_test(int n, double* delta_t, double3* area) {

	int index = threadIdx.x;
	int stride = blockDim.x;

	for (int i = index; i < n; i += stride) {
		double3 tmp;
		tmp = area[i];

		delta_t[i] = tmp.x + tmp.y + tmp.z;

	}
	return;
}



__global__ void clone_a_to_b(int n_cells, double4* a, double4* b) {

	//loop through cells

	int index = blockIdx.x * blockDim.x + threadIdx.x;;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < n_cells; i += stride) {

		if (i < n_cells) {
			double4 tmp1 = a[i];
			double4 tmp2;

			tmp2.x = tmp1.x;
			tmp2.y = tmp1.y;
			tmp2.z = tmp1.z;
			tmp2.w = tmp1.w;

			b[i] = tmp2;

		}
	}
}

__global__ void fill_zero(int n_cells, double* a) {

	//loop through cells

	int index = blockIdx.x * blockDim.x + threadIdx.x;;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < n_cells; i += stride) {
		if (i < n_cells) {
			a[i] = 0;

		}
	}
}

__global__ void fill_double(int n_cells, double* a, double val) {

	//loop through cells

	int index = blockIdx.x * blockDim.x + threadIdx.x;;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < n_cells; i += stride) {
		if (i < n_cells) {
			a[i] = val;

		}
	}
}




__global__ void square(int n_cells, double* a) {

	//loop through cells

	int index = blockIdx.x * blockDim.x + threadIdx.x;;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < n_cells; i += stride) {
		if (i < n_cells) {
			a[i] = a[i] * a[i];

		}
	}
}


__global__ void add(int n_cells, double* a, double *b) {

	//loop through cells

	int index = blockIdx.x * blockDim.x + threadIdx.x;;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < n_cells; i += stride) {
		if (i < n_cells) {
			a[i] = a[i] + b[i];

		}
	}
}